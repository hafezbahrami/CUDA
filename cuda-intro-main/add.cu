
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
    // Varibles to allow kernal loop to run in parallel
    // We can use special cuda variables to calculate which index of the array we are going to operate on
    // (current block id) * (number of threads in block) + (index of current thread in block)
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // stride is set to the entire size of the array, this is a little confusing but a blog about this
    // approach can be found here https://developer.nvidia.com/blog/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
    // The i < n condition in the loop statement will prevent calculations out of the array bounds
    int stride = blockDim.x * gridDim.x;  // (number of threads in block) * (number of blocks)

    // loop using special index and stride values
    for (int i = index; i < n; i+=stride)
        y[i] = x[i] + y[i];
}

int main(void)
{
    // Test Parameters
    int N = 1<<20; // 1M elements
    float *x, *y; //define x and y as floats

    // Tuning Variables 
    int threads_in_block = 256; // CUDA uses 32 threads/block so use multiples of 32
    int number_of_blocks = ( N + threads_in_block - 1 ) / threads_in_block; // Calculate number of blocks need to get "N" threads

    // Allocate Unified Memory – accessible from CPU or GPU as arrays
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run kernel on 1M elements on the GPU
    add<<<number_of_blocks, threads_in_block>>>(N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}